#include "hip/hip_runtime.h"
#include "CUDA.cuh"

__global__
void CUDA_MATRIX_MULTI(int m, int p, int n, int trash, double* A, double* B, double* C)
{
	int k = threadIdx.x + blockIdx.x * blockDim.x; //index of C

	if (k < m * n) {
		double temp = 0;
		for (int t = 0; t < p; t++) {
			temp += A[t + k / n * p] * B[k % n + t * n];
		}
		C[k] = temp;
	}
}
__global__
void CUDA_MATRIX_ADD_COLUMN(int m, int nC, int nA, int trash, double* A, double* B, double* C)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int mnC = m * nC;
	int nB = nC - nA;
	if (i < mnC)
	{
		int x = i / nC;
		int y = i % nC;
		if (y < nA)
			C[y + x * nC] = A[y + x * nA];
		else
			C[y + x * nC] = B[y - nA + x * nB];
	}
}
__global__
void CUDA_MATRIX_ADD_ROW(int mC, int nC, int mA, int trash, double* A, double* B, double* C)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int mnC = mC * nC;
	if (i < mnC)
	{
		int x = i / nC;
		int y = i % nC;
		if (x < mA)
			C[y + x * nC] = A[y + x * nC];
		else
			C[y + x * nC] = B[y + (x - mA) * nC];
	}
}
__global__
void CUDA_MATRIX_TRANS(int mC, int nC, int  mnC, int trash, double* A, double* trash2, double* C)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < mnC)
	{
		int x = i / nC;
		int y = i % nC;
		C[y + x * nC] = A[x + y * mC];
	}
}
__global__
void CUDA_MATRIX_CUT_COL(int mA, int nA, int start, int end, double* A, double* trahs, double* C)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int num = mA * nA;
	if (i < num)
	{
		int x = i / nA;
		int y = i % nA;
		if (y >= start && y <= end)
			C[y - start + x * (end - start + 1)] = A[i];
	}
}
__global__
void CUDA_MATRIX_CUT_ROW(int mA, int nA, int start, int end, double* A, double* trash, double* C)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int num = mA * nA;
	if (i < num)
	{
		int x = i / nA;
		int y = i % nA;
		if (x >= start && x <= end)
			C[y + (x - start) * nA] = A[i];
	}
}
__global__
void CUDA_MATRIX_COPY_COL(int mA, int nA, int nC, int end, double* A, double* trash, double* C)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int num = mA * nA;
	if (i < num)
	{	
		for (int k = 0; k <= end; k++)
		{
			C[k + i * nC] = A[i];
		}
	}
}
EMATRIXD* CUDA_MATRIX_CONVERT(EMATRIXD* A, EMATRIXD* B, int start, int end, const unsigned char FUN)
{
	double* CUDA_A = NULL;
	double* CUDA_B = NULL;
	double* CUDA_C = NULL;
	int size_A = 0, size_B = 0, size_C = 0;
	if (A != nullptr)
	{

		size_A = sizeof(double) * A->m * A->n;
		hipMalloc(&CUDA_A, size_A);
		hipMemcpy(CUDA_A, A->vectors, size_A, hipMemcpyHostToDevice);
	}
	if (B != nullptr)
	{

		size_B = sizeof(double) * B->m * B->n;
		hipMalloc(&CUDA_B, size_B);
		hipMemcpy(CUDA_B, B->vectors, size_B, hipMemcpyHostToDevice);

	}
	EMATRIXD* C = (EMATRIXD*)malloc(sizeof(EMATRIXD));
	void (*temp)(int, int, int, int, double*, double*, double*);
	long int bx = 0, by = 0, bz = 0, bi = 0;
	switch (FUN)
	{
	case MULTI:
		if (A->n != B->m)
		{
			elog << "[ERROR] MULTI MATRIX:" + to_string(A->n) + to_string(B->m) + "is not equal\n";
			return nullptr;
		}
		temp = CUDA_MATRIX_MULTI;
		size_C = A->m * B->n;
		C->m = A->m;
		C->n = B->n;
		bx = C->m;
		by = A->n;
		bz = C->n;
		break;
	case ADD_COL:
		if (A->m != B->m)
		{
			elog << "[ERROR] ADD COLUMN:" + to_string(A->m) + " : " + to_string(B->m) + "is not equal\n";
			return nullptr;
		}
		temp = CUDA_MATRIX_ADD_COLUMN;
		size_C = A->n * A->m + B->n * A->m;//for the speed
		C->m = A->m;
		C->n = A->n + B->n;
		bx = C->m;
		by = C->n;
		bz = A->n;
		break;
	case ADD_ROW:
		if (A->n != B->n)
		{
			elog << "[ERROR] ADD ROW:" + to_string(A->n) + " : " + to_string(B->n) + "is not equal\n";
			return nullptr;
		}
		temp = CUDA_MATRIX_ADD_ROW;
		size_C = A->n * A->m + B->n * A->m;//for the speed
		C->m = A->m + B->m;
		C->n = A->n;
		bx = C->m;
		by = C->n;
		bz = A->m;
		break;
	case TRANS:
		temp = CUDA_MATRIX_TRANS;
		size_C = A->n * A->m;//for the speed
		C->m = A->n;
		C->n = A->m;
		bx = C->m;
		by = C->n;
		bz = C->m * C->n;
		break;
	case CUT_COL:
		if (start <0 && end > A->n)
		{
			elog << "[ERROR] CUT COL:" + to_string(start) + " : " + to_string(end) + "is out of range\n";
			return nullptr;
		}
		temp = CUDA_MATRIX_CUT_COL;
		size_C = A->m * (end - start + 1);
		C->m = A->m;
		C->n = (end - start + 1);
		bx = A->m;
		by = A->n;
		bz = start;
		bi = end;
		break;
	case CUT_ROW:
		if (start <0 && end > A->m)
		{
			elog << "[ERROR] CUT ROW:" + to_string(start) + " : " + to_string(end) + "is out of range\n";
			return nullptr;
		}
		temp = CUDA_MATRIX_CUT_ROW;
		size_C = A->n * (end - start + 1);
		C->m = (end - start + 1);
		C->n = A->n;
		bx = A->m;
		by = A->n;
		bz = start;
		bi = end;
		break;
	case COPY_COL:
		temp = CUDA_MATRIX_COPY_COL;
		size_C = (A->n + end)* A->m;
		C->m = A->m;
		C->n = (A->n + end);
		bx = A->m;
		by = A->n;
		bz = C->n;
		bi = end;
		break;
	}

	hipMalloc(&CUDA_C, sizeof(double) * size_C);
	dim3 blockPerGrid(512, 1, 1);
	dim3 threadsPerBlock(512, 1, 1);

	temp << <blockPerGrid, threadsPerBlock >> > (bx, by, bz, bi, CUDA_A, CUDA_B, CUDA_C);

	C->vectors = (double*)malloc(sizeof(double) * size_C);
	hipMemcpy(C->vectors, CUDA_C, sizeof(double) * size_C, hipMemcpyDeviceToHost);
	if (CUDA_A != NULL)
		hipFree(CUDA_A);
	if (CUDA_B != NULL)
		hipFree(CUDA_B);
	if (CUDA_C != NULL)
		hipFree(CUDA_C);
	return C;
}